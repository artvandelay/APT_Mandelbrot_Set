
#include <hip/hip_runtime.h>
/* 
 * File:   Complex.cu
 * 
 * Created on June 24, 2012
 * 
 * Purpose:  Simple complex number class for use on GPU
 * 
 * If it works, it was written by Brian Swenson.
 * Otherwise, I have no idea who wrote it.
 */

class Complex 
{
public:
    float   r;
    float   i;
    Complex( float a, float b ) : r(a), i(b)  {}
    Complex(const Complex& x) : r(x.r), i(x.i) {}
    float magnitude2( void ) {
        return r * r + i * i;
    }
    Complex operator*(const Complex& a) {
        return Complex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    Complex operator+(const Complex& a) {
        return Complex(r+a.r, i+a.i);
    }
};
